#include "hip/hip_runtime.h"

#include "nOCTcudaDLLHeader.cuh"

// status and control parameters
static int gnMode = -1; 
static int gnAllocationStatus = 0; 
static bool gbIsReferenceRecorded = false; 
static bool gbIsCalibrationLoaded = false; 
static bool gbIsDispersionLoaded = false; 


static int gnRawLineLength;
static int gnRawNumberLines;            // number of lines in a frame 
// static int gnCalibrationNumberLines;
static int gnProcessNumberLines;        // number of lines in a chunk
static int gnProcessedNumberLines;
static int gnPerpendicular;
static int gnMidLength;

/* raw spectra arrays */
    // common
static short* d_gpnRawIMAQ;             // device: raw spectra from camera
static float* d_gpfRawIMAQ;             // device: raw spectra (gpfRawCalibration) 
static float* gpfIMAQPitched;           // device: raw spectra copied to pitched memory (gpfProcessCalibration)
static size_t gnIMAQPitch;              // gnProcessCalibrationPitch
    // PS-SD-OCT
static short* d_gpnRawIMAQParallel;     // device: raw spectra from camera
static float* d_gpfRawIMAQParallel;     // device: raw spectra (gpfRawCalibration) 
static short* d_gpnRawIMAQPerpendicular;    // device: raw spectra from camera
static float* d_gpfRawIMAQPerpendicular;    // device: raw spectra (gpfRawCalibration) 


/* reference */
    // common 
static float* gpfReference; 
    // PS-SD-OCT
static float* gpfReferenceParallel;
static float* gpfReferencePerpendicular;

/* fft */ 
static hipfftHandle gchForward;
static hipfftComplex* gpcProcessDepthProfile; 
static size_t gnProcessDepthProfilePitch;    


// calibration mask
static int gnCalibrationStart;
static int gnCalibrationStop;
static int gnCalibrationRound;
static float* gpfCalibrationMask;

// reverse fft
static hipfftComplex* gpcProcessSpectrum;
static size_t gnProcessSpectrumPitch;
static hipfftHandle gchReverse;

// phase
static float* gpfProcessPhase;
static size_t gnProcessPhasePitch;

// unwrap
static float gfPiEps = (float)(acos(-1.0) - 1.0e-30);
static float gf2Pi = (float)(2.0 * acos(-1.0));

// linear fit and interpolation
static float* gpfLeftPhase;
static float* gpfRightPhase;
static float* gpfKLineCoefficients;
static float* gpfProcessK;
static size_t gnKPitch;
static int* gpnProcessIndex;
static size_t gnIndexPitch;
static int* gpnProcessAssigned;
static size_t gnAssignedPitch;
static int gnKMode;
static float* gpfProcessSpectrumK;
static size_t gnSpectrumKPitch;

static float* gpfProcessOCT;
static size_t gnProcessOCTPitch;
static hipfftComplex* gpcProcessedOCT;

// dispersion mask
static int gnDispersionStart;
static int gnDispersionStop;
static int gnDispersionRound;
static float* gpfDispersionMask;

// dispersion correction
static hipfftComplex* gpcDispersionCorrection;
static hipfftHandle gchForwardComplex;
static hipfftComplex* gpcProcessKCorrected;
static size_t gnKCorrectedPitch;


int getDeviceCount(int* nNumberDevices) {
	
    // check for GPU
	int nDevices = -1; 
	int nRet = hipGetDeviceCount(&nDevices); 
	if (nRet == hipSuccess)
		*(nNumberDevices) = nDevices; 
	return nRet; 

}

int getDeviceName(int nDeviceNumber, char* strDeviceName) {
    
    // check for GPU
    hipDeviceProp_t currentDevice;
    int nRet = hipGetDeviceProperties(&currentDevice, nDeviceNumber);
    if (nRet == hipSuccess) {
        sprintf(strDeviceName, "%s (%d SMs, %d b/s, %d t/b, %d t/s, %d shared kB, %d GB)",
            currentDevice.name,
            currentDevice.multiProcessorCount,
            currentDevice.maxBlocksPerMultiProcessor,
            currentDevice.maxThreadsPerBlock,
            currentDevice.maxThreadsPerMultiProcessor,
            currentDevice.sharedMemPerBlock / 1024,
            currentDevice.totalGlobalMem / 1024 / 1024 / 1024);

    }	// if (nRet
    return nRet; 

}

int initialize(int nMode, int nRawLineLength, int nRawNumberLines, int nProcessNumberLines, int nProcessedNumberLines) {
    
    cleanup(nMode);

    // copy parameters to global parameters
    gnMode = nMode;
    gnRawLineLength = nRawLineLength;
    gnRawNumberLines = nRawNumberLines;                 // number of lines in a frame
    gnProcessNumberLines = nProcessNumberLines;         // number of lines in a chunk
    gnProcessedNumberLines = nProcessedNumberLines;

    int nActualProcessNumberLines; 

    // allocate memory  

    switch (nMode) {
    case 0: // SD-OCT
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = 1;
        nActualProcessNumberLines = gnProcessNumberLines; 

        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));

        gpuErrchk(hipMalloc((void**)&gpfReference, gnRawLineLength * sizeof(float))); 

        break;
    case 1: // PS SD-OCT
        gnPerpendicular = 1;
        // gnCalibrationNumberLines = gnRawNumberLines; // QUESTION: what is this parameter?
        nActualProcessNumberLines = gnProcessNumberLines >> 1;      // only process every other line in each array

        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));
        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));
        
        gpuErrchk(hipMalloc((void**)&gpfReferenceParallel, gnRawLineLength * sizeof(float)));
        gpuErrchk(hipMalloc((void**)&gpfReferencePerpendicular, gnRawLineLength * sizeof(float)));        

        break;
    case 2: // line field
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = 1;
        break;
    case 3: // OFDI
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = gnRawNumberLines;
        break;
    case 4: // PS OFDI
        gnPerpendicular = 1;
        // gnCalibrationNumberLines = gnRawNumberLines;
        break;
    } // switch (nMode)

    gpuErrchk(hipMallocPitch((void**)&gpfIMAQPitched, &gnIMAQPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
     
    gnMidLength = (int)(gnRawLineLength / 2 + 1);
    gpuErrchk(hipMallocPitch((void**)&gpcProcessDepthProfile, &gnProcessDepthProfilePitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));
    int nRank = 1;
    int pn[] = { gnRawLineLength };
    int nIStride = 1, nOStride = 1;
    int nIDist = gnIMAQPitch / sizeof(float);
    int nODist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    int pnINEmbed[] = { 0 };
    int pnONEmbed[] = { 0 };
    int nBatch = gnProcessNumberLines >> 1;
    hipfftPlanMany(&gchForward, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_R2C, nBatch);

    gpuErrchk(hipMalloc((void**)&gpfCalibrationMask, gnRawLineLength * sizeof(float)));

    gpuErrchk(hipMallocPitch((void**)&gpcProcessSpectrum, &gnProcessSpectrumPitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));
    nIDist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    nODist = gnProcessSpectrumPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchReverse, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gpuErrchk(hipMallocPitch((void**)&gpfProcessPhase, &gnProcessPhasePitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));

    hipMalloc((void**)&gpfLeftPhase, sizeof(float));
    hipMalloc((void**)&gpfRightPhase, sizeof(float));
    hipMalloc((void**)&gpfKLineCoefficients, 2 * sizeof(float));
    gpuErrchk(hipMallocPitch((void**)&gpfProcessK, &gnKPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessIndex, &gnIndexPitch, gnRawLineLength * sizeof(int), nActualProcessNumberLines));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessAssigned, &gnAssignedPitch, gnRawLineLength * sizeof(int), nActualProcessNumberLines));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessSpectrumK, &gnSpectrumKPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessOCT, &gnProcessOCTPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
    gpuErrchk(hipHostMalloc((void**)&gpcProcessedOCT, (gnMidLength * gnProcessedNumberLines) * sizeof(hipfftComplex)));

    gpuErrchk(hipMalloc((void**)&gpfDispersionMask, gnRawLineLength * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&gpcDispersionCorrection, gnRawLineLength * sizeof(hipfftComplex)));
    gpuErrchk(hipMallocPitch((void**)&gpcProcessKCorrected, &gnKCorrectedPitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));

    nIDist = gnKCorrectedPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchForwardComplex, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gpuErrchk(hipDeviceSynchronize());     // QUESTION: will hipDeviceSynchronize slow down the performance?

    gnAllocationStatus = 1;

    return -1;

} // int initialize

int cleanup(int nMode) {

    // free memory allocations
    if (gnAllocationStatus == 1) {
        
        
        switch (nMode)
        {
        case 0: // SD-OCT
            // gpuErrchk(hipHostFree(h_gpnRawIMAQ));
            gpuErrchk(hipFree(d_gpnRawIMAQ));
            gpuErrchk(hipFree(d_gpfRawIMAQ));

            gpuErrchk(hipFree(gpfReference)); 
            
            break;
        case 1: // PS SD-OCT 
            // gpuErrchk(hipHostFree(h_gpnRawIMAQParallel));
            gpuErrchk(hipFree(d_gpnRawIMAQParallel));
            gpuErrchk(hipFree(d_gpfRawIMAQParallel));
            // gpuErrchk(hipHostFree(h_gpnRawIMAQPerpendicular));
            gpuErrchk(hipFree(d_gpnRawIMAQPerpendicular));
            gpuErrchk(hipFree(d_gpfRawIMAQPerpendicular));

            gpuErrchk(hipFree(gpfReferenceParallel));
            gpuErrchk(hipFree(gpfReferencePerpendicular));

            break;
        case 2: // line field
            
            break;
        case 3: // OFDI
            
            break;
        case 4: // PS OFDI
            
            break;
            break;
        }
        
        gpuErrchk(hipFree(gpfIMAQPitched));   
        gpuErrchk(hipFree(gpcProcessDepthProfile));
        hipfftDestroy(gchForward);
        gpuErrchk(hipFree(gpfCalibrationMask));
        gpuErrchk(hipFree(gpcProcessSpectrum));
        hipfftDestroy(gchReverse);
        gpuErrchk(hipFree(gpfProcessPhase));
        hipFree(gpfLeftPhase);
        hipFree(gpfRightPhase);
        hipFree(gpfKLineCoefficients);
        hipFree(gpfProcessK);
        hipFree(gpnProcessIndex);
        hipFree(gpnProcessAssigned);
        hipFree(gpfProcessSpectrumK);
        hipFree(gpfProcessOCT);
        hipHostFree(gpcProcessedOCT);
        gpuErrchk(hipFree(gpfDispersionMask));
        gpuErrchk(hipFree(gpcDispersionCorrection));
        hipfftDestroy(gchForwardComplex);
        hipFree(gpcProcessKCorrected);

        gnAllocationStatus = 0;
    }   // if (gnAllocationStatus
    return -1;

}

int getReferenceData(int nMode, short* pnReferenceParallel, short* pnReferencePerpendicular, bool bIsReferenceRecorded) {
    // copy parameters to global parameters
    gbIsReferenceRecorded = bIsReferenceRecorded; 

    if (bIsReferenceRecorded == true) {        

        switch (nMode) {
        case 0: // SD-OCT
            // data type conversion (on host)
            float* pfReference = new float[gnRawLineLength];
            for (int i; i < gnRawLineLength; i++) {
                pfReference[i] = (float)pnReferenceParallel[i];
            }

            // copy data to device
            gpuErrchk(hipMemcpy(gpfReference, pfReference, gnRawLineLength * sizeof(short), hipMemcpyHostToDevice));
            gpuErrchk(hipDeviceSynchronize());

            delete[] pfReference;

            break;
        case 1: // PS SD-OCT
            // data type conversion (on host)
            float* pfReferenceParallel = new float[gnRawLineLength];
            float* pfReferencePerpendicular = new float[gnRawLineLength];
            for (int i; i < gnRawLineLength; i++) {
                pfReferenceParallel[i] = (float)pnReferenceParallel[i];
                pfReferencePerpendicular[i] = (float)pnReferencePerpendicular[i];
            }

            // copy data to device
            gpuErrchk(hipMemcpy(gpfReferenceParallel, pfReferenceParallel, gnRawLineLength * sizeof(short), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(gpfReferencePerpendicular, pfReferencePerpendicular, gnRawLineLength * sizeof(short), hipMemcpyHostToDevice));
            gpuErrchk(hipDeviceSynchronize());

            delete[] pfReferenceParallel;
            delete[] pfReferencePerpendicular; 

            break;
        case 2: // line field

            break;
        case 3: // OFDI

            break;
        case 4: // PS OFDI

            break;
        } // switch (nMode)

    }

    return -1; 
}

int getDataSDOCT(void* pnIMAQ) {

    


    return -1; 
}

int getDataPSSDOCT(void* pnIMAQParallel, void* pnIMAQPerpendicular) {
    // copy data to device
    gpuErrchk(hipMemcpy(d_gpnRawIMAQParallel, pnIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short), hipMemcpyHostToDevice)); 
    gpuErrchk(hipMemcpy(d_gpnRawIMAQPerpendicular, pnIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    return -1; 
}


int processSDOCT() {
    return -1; 

}


int processPSSDOCT() {

    int nThreadsPerBlock; 
    dim3 d3Threads; 
    dim3 d3Blocks; 

    // convert to float type 
    d3Threads.x = 512;  d3Threads.y = 1;    d3Threads.z = 1; 
    d3Blocks.x = (gnRawLineLength * gnRawNumberLines - 1) / d3Threads.x + 1; 
    d3Blocks.y = 1;     d3Blocks.z = 1;
    convert2Float << <d3Blocks, d3Threads >> > (d_gpnRawIMAQParallel, d_gpfRawIMAQParallel, gnRawLineLength * gnRawNumberLines); 
    gpuErrchk(hipPeekAtLastError());
    convert2Float << <d3Blocks, d3Threads >> > (d_gpnRawIMAQPerpendicular, d_gpfRawIMAQPerpendicular, gnRawLineLength * gnRawNumberLines);
    gpuErrchk(hipPeekAtLastError()); 

    gpuErrchk(hipDeviceSynchronize()); 

    // loop through cameras
    for (int nCam = 0; nCam < 2; nCam++) {  // nCam = 0: parallel camera; nCam = 1: perpendicular camera
        int nNumberLinesPerChunk = gnProcessNumberLines;    // value set in C# UI
        int nNumberChunks = (gnRawNumberLines - 1) / gnProcessNumberLines + 1;  // QUESTION: need to double check. why previous method?

        // loop through chunks
        for (int nChunk = 0; nChunk < nNumberChunks; nChunk++) { 

            // loop through even and odd lines, respectively
            int nSrcPtrOffset = nChunk * (gnRawLineLength * nNumberLinesPerChunk); 
            for (int nOddEven = 0; nOddEven < 2; nOddEven++) { // nOddEven = 0: process even lines; nOddEven = 1; process odd lines
                // copy a chunk: in each data array (on device now), copy every other line 
                switch (nOddEven) {
                case 0: // even lines 
                    gpuErrchk(hipMemcpy2D(gpfIMAQPitched, gnIMAQPitch, d_gpfRawIMAQ + nSrcPtrOffset, 2 * gnIMAQPitch, gnIMAQPitch, nNumberLinesPerChunk >> 1, hipMemcpyDeviceToDevice));
                    break; 
                case 1: // odd lines
                    gpuErrchk(hipMemcpy2D(gpfIMAQPitched, gnIMAQPitch, d_gpfRawIMAQ + gnRawLineLength + nSrcPtrOffset, 2 * gnIMAQPitch, gnIMAQPitch, nNumberLinesPerChunk >> 1, hipMemcpyDeviceToDevice)); 
                    break;
                } // switch (nOddEven)
                
                gpuErrchk(hipDeviceSynchronize()); 

                /* reference */ 
                if (gbIsReferenceRecorded == false) { // no reference data recorded
                    // calculate reference 
                    d3Threads.x = 128;
                    d3Threads.y = 1024 / d3Threads.x;
                    d3Threads.z = 1;
                    d3Blocks.x = nNumberLinesPerChunk / d3Threads.x;
                    d3Blocks.y = 1;
                    d3Blocks.z = 1;

                        // different cameras
                    switch (nCam) {
                    case 0: // parallel camera
                        calculateMean << <d3Blocks, d3Threads >> > (gpfIMAQPitched, gpfReferenceParallel, nNumberLinesPerChunk >> 1, gnRawLineLength);
                        break;
                    case 1: // perpendicular camera
                        calculateMean << <d3Blocks, d3Threads >> > (gpfIMAQPitched, gpfReferencePerpendicular, nNumberLinesPerChunk >> 1, gnRawLineLength);
                        break; 
                    }
                    gpuErrchk(hipPeekAtLastError()); 
                } // if (gbIsReferenceRecorded == false)
                
                // subtract reference 
                d3Threads.x = 32;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = nNumberLinesPerChunk / d3Threads.x;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                    // different cameras
                switch (nCam) {
                case 0: // parallel camera
                    subtractMean << <d3Blocks, d3Threads >> > (gpfIMAQPitched, gpfReferenceParallel, nNumberLinesPerChunk >> 1, gnRawLineLength);
                    break;
                case 1: // perpendicular camera
                    subtractMean << <d3Blocks, d3Threads >> > (gpfIMAQPitched, gpfReferencePerpendicular, nNumberLinesPerChunk >> 1, gnRawLineLength);
                    break;
                }
                gpuErrchk(hipPeekAtLastError()); 

                /* forward fft */
                gpuErrchk(hipMemset2D(gpcProcessDepthProfile, gnProcessDepthProfilePitch, 0.0, gnProcessDepthProfilePitch, nNumberLinesPerChunk >> 1));
                hipfftExecR2C(gchForward, gpfIMAQPitched, gpcProcessDepthProfile);

                /* mask */
                // calculate mask: QUESTION can be done in CPU in the initialize function? (small data size, avoid warp divergence) 
                nThreadsPerBlock = 512;
                calculateMask << <gnRawLineLength / nThreadsPerBlock, nThreadsPerBlock >> > (gpfCalibrationMask, gnRawLineLength, 50, 100, 16);     // grab these numbers from C# UI

                // apply mask
                d3Threads.x = 32;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = gnProcessNumberLines / d3Threads.x;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                applyMask << <d3Blocks, d3Threads >> > (gpcProcessDepthProfile, gpfCalibrationMask, nNumberLinesPerChunk >> 1, gnRawLineLength);
                gpuErrchk(hipPeekAtLastError());

                /* reverse fft */
                hipfftExecC2C(gchReverse, gpcProcessDepthProfile, gpcProcessSpectrum, HIPFFT_BACKWARD);

                /* calculate phase */
                d3Threads.x = 32;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = gnRawLineLength / d3Threads.x;
                d3Blocks.y = (gnProcessNumberLines >> 1) / d3Threads.y;
                d3Blocks.z = 1;
                calculatePhase << <d3Blocks, d3Threads >> > (gpcProcessSpectrum, gpfProcessPhase, nNumberLinesPerChunk >> 1, gnRawLineLength);
                gpuErrchk(hipPeekAtLastError());

                d3Threads.x = 256;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = (nNumberLinesPerChunk >> 1) / d3Threads.y;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                unwrapPhase << <d3Blocks, d3Threads >> > (gpfProcessPhase, nNumberLinesPerChunk >> 1, gnRawLineLength, gfPiEps, gf2Pi);

                d3Threads.x = 256;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = (nNumberLinesPerChunk >> 1) / d3Threads.y;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                matchPhase << <d3Blocks, d3Threads >> > (gpfProcessPhase, nNumberLinesPerChunk >> 1, gnRawLineLength, gf2Pi);

                nThreadsPerBlock = 32;
                getPhaseLimits << <2, nThreadsPerBlock >> > (gpfProcessPhase, nNumberLinesPerChunk >> 1, gnRawLineLength, 32, gnRawLineLength - 32, gpfLeftPhase, gpfRightPhase);

                gnKMode = 1;
                d3Threads.x = 128;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = (nNumberLinesPerChunk >> 1) / d3Threads.y;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                hipMemset2D(gpnProcessAssigned, gnAssignedPitch, 0, gnRawLineLength * sizeof(int), nNumberLinesPerChunk >> 1);
                calculateK << <d3Blocks, d3Threads >> > (gpfProcessPhase, gpfProcessK, gpnProcessAssigned, gpnProcessIndex, nNumberLinesPerChunk >> 1, gnRawLineLength, gpfKLineCoefficients, 32, gnRawLineLength - 32, gpfLeftPhase, gpfRightPhase, gnKMode);

                d3Threads.x = 128;
                d3Threads.y = 1024 / d3Threads.x;
                d3Threads.z = 1;
                d3Blocks.x = (nNumberLinesPerChunk >> 1) / d3Threads.y;
                d3Blocks.y = 1;
                d3Blocks.z = 1;
                cleanIndex << <d3Blocks, d3Threads >> > (gpfProcessK, gpnProcessIndex, gpnProcessAssigned, nNumberLinesPerChunk >> 1, gnRawLineLength);





            } // for (int nOddEven = 0; nOddEven < 2; nOddEven++)
             


        } // for (int nChunk = 0; nChunk < nNumberChunks; nChunk++)


    } // for (int nCam = 0; nCam < 2; nCam++)




    return -1; 
}
