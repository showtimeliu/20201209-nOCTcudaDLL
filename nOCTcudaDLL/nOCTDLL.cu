#include "hip/hip_runtime.h"

#include "nOCTcudaDLLHeader.cuh"

static int gnMode = -1;
static int gnRawLineLength;
static int gnRawNumberLines;            // number of lines in a frame 
// static int gnCalibrationNumberLines;
static int gnProcessNumberLines;        // number of lines in a chunk
static int gnProcessedNumberLines;
static int gnPerpendicular;
static int gnAllocationStatus = 0;
static int gnMidLength;

/* raw spectra arrays */
    // common
// static short* h_gpnRawIMAQ;             // host: raw spectra from camera 
static short* d_gpnRawIMAQ;             // device: raw spectra from camera
static float* d_gpfRawIMAQ;             // device: raw spectra (gpfRawCalibration) 
static float* gpfIMAQPitched;           // device: raw spectra copied to pitched memory (gpfProcessCalibration)
static size_t gnIMAQPitch;              // gnProcessCalibrationPitch
    // PS-SD-OCT
// static short* h_gpnRawIMAQParallel;     // host: raw spectra from camera 
static short* d_gpnRawIMAQParallel;     // device: raw spectra from camera
static float* d_gpfRawIMAQParallel;     // device: raw spectra (gpfRawCalibration) 
// static short* h_gpnRawIMAQPerpendicular;    // host: raw spectra from camera 
static short* d_gpnRawIMAQPerpendicular;    // device: raw spectra from camera
static float* d_gpfRawIMAQPerpendicular;    // device: raw spectra (gpfRawCalibration) 


/* reference */
    // common 
static float* gpfReference; 
    // PS-SD-OCT
static float* gpfReferenceEven;
static float* gpfReferenceOdd;

/* fft */ 
static hipfftHandle gchForward;
static hipfftComplex* gpcProcessDepthProfile; 
static size_t gnProcessDepthProfilePitch;    


// calibration mask
static int gnCalibrationStart;
static int gnCalibrationStop;
static int gnCalibrationRound;
static float* gpfCalibrationMask;

// reverse fft
static hipfftComplex* gpcProcessSpectrum;
static size_t gnProcessSpectrumPitch;
static hipfftHandle gchReverse;

// phase
static float* gpfProcessPhase;
static size_t gnProcessPhasePitch;

// unwrap
static float gfPiEps = (float)(acos(-1.0) - 1.0e-30);
static float gf2Pi = (float)(2.0 * acos(-1.0));

// linear fit and interpolation
static float* gpfLeftPhase;
static float* gpfRightPhase;
static float* gpfKLineCoefficients;
static float* gpfProcessK;
static size_t gnKPitch;
static int* gpnProcessIndex;
static size_t gnIndexPitch;
static int* gpnProcessAssigned;
static size_t gnAssignedPitch;
static int gnKMode;
static float* gpfProcessSpectrumK;
static size_t gnSpectrumKPitch;

static float* gpfProcessOCT;
static size_t gnProcessOCTPitch;
static hipfftComplex* gpcProcessedOCT;

// dispersion mask
static int gnDispersionStart;
static int gnDispersionStop;
static int gnDispersionRound;
static float* gpfDispersionMask;

// dispersion correction
static hipfftComplex* gpcDispersionCorrection;
static hipfftHandle gchForwardComplex;
static hipfftComplex* gpcProcessKCorrected;
static size_t gnKCorrectedPitch;


int getDeviceCount(int* nNumberDevices) {
	
    // check for GPU
	int nDevices = -1; 
	int nRet = hipGetDeviceCount(&nDevices); 
	if (nRet == hipSuccess)
		*(nNumberDevices) = nDevices; 
	return nRet; 

}

int getDeviceName(int nDeviceNumber, char* strDeviceName) {
    
    // check for GPU
    hipDeviceProp_t currentDevice;
    int nRet = hipGetDeviceProperties(&currentDevice, nDeviceNumber);
    if (nRet == hipSuccess) {
        sprintf(strDeviceName, "%s (%d SMs, %d b/s, %d t/b, %d t/s, %d shared kB, %d GB)",
            currentDevice.name,
            currentDevice.multiProcessorCount,
            currentDevice.maxBlocksPerMultiProcessor,
            currentDevice.maxThreadsPerBlock,
            currentDevice.maxThreadsPerMultiProcessor,
            currentDevice.sharedMemPerBlock / 1024,
            currentDevice.totalGlobalMem / 1024 / 1024 / 1024);

    }	// if (nRet
    return nRet; 

}

int initialize(int nMode, int nRawLineLength, int nRawNumberLines, int nProcessNumberLines, int nProcessedNumberLines) {
    
    cleanup(nMode);

    // copy parameters to global parameters
    gnMode = nMode;
    gnRawLineLength = nRawLineLength;
    gnRawNumberLines = nRawNumberLines;                 // number of lines in a frame
    gnProcessNumberLines = nProcessNumberLines;         // number of lines in a chunk
    gnProcessedNumberLines = nProcessedNumberLines;

    int nActualProcessNumberLines; 

    // allocate memory  

    switch (nMode) {
    case 0: // SD-OCT
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = 1;
        nActualProcessNumberLines = gnProcessNumberLines; 

        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQ, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));

        gpuErrchk(hipMalloc((void**)&gpfReference, gnRawLineLength * sizeof(float))); 

        break;
    case 1: // PS SD-OCT
        gnPerpendicular = 1;
        // gnCalibrationNumberLines = gnRawNumberLines; // ???
        nActualProcessNumberLines = gnProcessNumberLines >> 1;      // only process odd or even lines

        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));
        // gpuErrchk(hipHostMalloc((void**)&h_gpnRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpnRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short)));
        gpuErrchk(hipMalloc((void**)&d_gpfRawIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(float)));
        
        gpuErrchk(hipMalloc((void**)&gpfReferenceEven, gnRawLineLength * sizeof(float)));
        gpuErrchk(hipMalloc((void**)&gpfReferenceOdd, gnRawLineLength * sizeof(float)));        

        break;
    case 2: // line field
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = 1;
        break;
    case 3: // OFDI
        gnPerpendicular = 0;
        // gnCalibrationNumberLines = gnRawNumberLines;
        break;
    case 4: // PS OFDI
        gnPerpendicular = 1;
        // gnCalibrationNumberLines = gnRawNumberLines;
        break;
    } // switch (nMode)

    gpuErrchk(hipMallocPitch((void**)&gpfIMAQPitched, &gnIMAQPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
     
    gnMidLength = (int)(gnRawLineLength / 2 + 1);
    gpuErrchk(hipMallocPitch((void**)&gpcProcessDepthProfile, &gnProcessDepthProfilePitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));
    int nRank = 1;
    int pn[] = { gnRawLineLength };
    int nIStride = 1, nOStride = 1;
    int nIDist = gnIMAQPitch / sizeof(float);
    int nODist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    int pnINEmbed[] = { 0 };
    int pnONEmbed[] = { 0 };
    int nBatch = gnProcessNumberLines >> 1;
    hipfftPlanMany(&gchForward, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_R2C, nBatch);

    gpuErrchk(hipMalloc((void**)&gpfCalibrationMask, gnRawLineLength * sizeof(float)));

    gpuErrchk(hipMallocPitch((void**)&gpcProcessSpectrum, &gnProcessSpectrumPitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));
    nIDist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    nODist = gnProcessSpectrumPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchReverse, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gpuErrchk(hipMallocPitch((void**)&gpfProcessPhase, &gnProcessPhasePitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));

    hipMalloc((void**)&gpfLeftPhase, sizeof(float));
    hipMalloc((void**)&gpfRightPhase, sizeof(float));
    hipMalloc((void**)&gpfKLineCoefficients, 2 * sizeof(float));
    gpuErrchk(hipMallocPitch((void**)&gpfProcessK, &gnKPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessIndex, &gnIndexPitch, gnRawLineLength * sizeof(int), nActualProcessNumberLines));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessAssigned, &gnAssignedPitch, gnRawLineLength * sizeof(int), nActualProcessNumberLines));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessSpectrumK, &gnSpectrumKPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessOCT, &gnProcessOCTPitch, gnRawLineLength * sizeof(float), nActualProcessNumberLines));
    gpuErrchk(hipHostMalloc((void**)&gpcProcessedOCT, (gnMidLength * gnProcessedNumberLines) * sizeof(hipfftComplex)));

    gpuErrchk(hipMalloc((void**)&gpfDispersionMask, gnRawLineLength * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&gpcDispersionCorrection, gnRawLineLength * sizeof(hipfftComplex)));
    gpuErrchk(hipMallocPitch((void**)&gpcProcessKCorrected, &gnKCorrectedPitch, gnRawLineLength * sizeof(hipfftComplex), nActualProcessNumberLines));

    nIDist = gnKCorrectedPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchForwardComplex, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gpuErrchk(hipDeviceSynchronize());

    gnAllocationStatus = 1;

    return -1;

} // int initialize

int cleanup(int nMode) {

    // free memory allocations
    if (gnAllocationStatus == 1) {
        
        
        switch (nMode)
        {
        case 0: // SD-OCT
            gpuErrchk(hipHostFree(h_gpnRawIMAQ));
            gpuErrchk(hipFree(d_gpnRawIMAQ));
            gpuErrchk(hipFree(d_gpfRawIMAQ));

            gpuErrchk(hipFree(gpfReference)); 
            
            break;
        case 1: // PS SD-OCT 
            gpuErrchk(hipHostFree(h_gpnRawIMAQParallel));
            gpuErrchk(hipFree(d_gpnRawIMAQParallel));
            gpuErrchk(hipFree(d_gpfRawIMAQParallel));
            gpuErrchk(hipHostFree(h_gpnRawIMAQPerpendicular));
            gpuErrchk(hipFree(d_gpnRawIMAQPerpendicular));
            gpuErrchk(hipFree(d_gpfRawIMAQPerpendicular));

            gpuErrchk(hipFree(gpfReferenceEven));
            gpuErrchk(hipFree(gpfReferenceOdd));

            break;
        case 2: // line field
            
            break;
        case 3: // OFDI
            
            break;
        case 4: // PS OFDI
            
            break;
            break;
        }
        
        gpuErrchk(hipFree(gpfIMAQPitched));   
        gpuErrchk(hipFree(gpcProcessDepthProfile));
        hipfftDestroy(gchForward);
        gpuErrchk(hipFree(gpfCalibrationMask));
        gpuErrchk(hipFree(gpcProcessSpectrum));
        hipfftDestroy(gchReverse);
        gpuErrchk(hipFree(gpfProcessPhase));
        hipFree(gpfLeftPhase);
        hipFree(gpfRightPhase);
        hipFree(gpfKLineCoefficients);
        hipFree(gpfProcessK);
        hipFree(gpnProcessIndex);
        hipFree(gpnProcessAssigned);
        hipFree(gpfProcessSpectrumK);
        hipFree(gpfProcessOCT);
        hipHostFree(gpcProcessedOCT);
        gpuErrchk(hipFree(gpfDispersionMask));
        gpuErrchk(hipFree(gpcDispersionCorrection));
        hipfftDestroy(gchForwardComplex);
        hipFree(gpcProcessKCorrected);

        gnAllocationStatus = 0;
    }   // if (gnAllocationStatus
    return -1;

}

int getDataSDOCT(void* pnIMAQ) {

    


    return -1; 
}

int getDataPSSDOCT(void* pnIMAQParallel, void* pnIMAQPerpendicular) {
    hipMemcpy(d_gpnRawIMAQParallel, pnIMAQParallel, (gnRawLineLength * gnRawNumberLines) * sizeof(short), hipMemcpyHostToDevice); 
    hipMemcpy(d_gpnRawIMAQPerpendicular, pnIMAQPerpendicular, (gnRawLineLength * gnRawNumberLines) * sizeof(short), hipMemcpyHostToDevice);

    gpuErrchk(hipDeviceSynchronize());

    return -1; 
}


int processSDOCT() {
    return -1; 

}


int processPSSDOCT() {

    int nThreadsPerBlock; 
    dim3 d3Threads; 
    dim3 d3Blocks; 

    // convert to float type 
    d3Threads.x = 512;  d3Threads.y = 1;    d3Threads.z = 1; 
    d3Blocks.x = gnRawLineLength * gnRawNumberLines / d3Threads.x; 
    d3Blocks.y = 1;     d3Blocks.z = 1;
    convert2Float << <d3Blocks, d3Threads >> > (d_gpnRawIMAQParallel, d_gpfRawIMAQParallel, gnRawLineLength * gnRawNumberLines); 
    gpuErrchk(hipPeekAtLastError());
    convert2Float << <d3Blocks, d3Threads >> > (d_gpnRawIMAQPerpendicular, d_gpfRawIMAQPerpendicular, gnRawLineLength * gnRawNumberLines);
    gpuErrchk(hipPeekAtLastError()); 

    // loop through cameras
    for (int nCam = 0; nCam < 2; nCam++) {

    }




    return -1; 
}
