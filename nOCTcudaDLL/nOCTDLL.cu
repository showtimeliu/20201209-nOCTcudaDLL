
#include "nOCTcudaDLLHeader.cuh"

int getDeviceCount(int* nNumberDevices) {
	// check for GPU
	int nDevices = -1; 
	int nRet = hipGetDeviceCount(&nDevices); 
	if (nRet == hipSuccess)
		*(nNumberDevices) = nDevices; 
	return nRet; 
}

int getDeviceName(int nDeviceNumber, char* strDeviceName) {
    // check for GPU
    hipDeviceProp_t currentDevice;
    int nRet = hipGetDeviceProperties(&currentDevice, nDeviceNumber);
    if (nRet == hipSuccess) {
        sprintf(strDeviceName, "%s (%d SMs, %d b/s, %d t/b, %d t/s, %d shared kB, %d GB)",
            currentDevice.name,
            currentDevice.multiProcessorCount,
            currentDevice.maxBlocksPerMultiProcessor,
            currentDevice.maxThreadsPerBlock,
            currentDevice.maxThreadsPerMultiProcessor,
            currentDevice.sharedMemPerBlock / 1024,
            currentDevice.totalGlobalMem / 1024 / 1024 / 1024);

    }	// if (nRet
    return nRet;
}