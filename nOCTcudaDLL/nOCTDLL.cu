#include "hip/hip_runtime.h"

#include "nOCTcudaDLLHeader.cuh"

static int gnMode = -1;
static int gnRawLineLength;
static int gnRawNumberLines;
static int gnCalibrationNumberLines;
static int gnProcessNumberLines;
static int gnProcessedNumberLines;
static int gnPerpendicular;
static int gnAllocationStatus = 0;
static int gnMidLength;

static float* gpfRawCalibration;
static float* gpfProcessCalibration;
static size_t gnProcessCalibrationPitch;

// reference
static float* gpfReferenceEven;
static float* gpfReferenceOdd;

// fft
static hipfftComplex* gpcProcessDepthProfile;
static size_t gnProcessDepthProfilePitch;
static hipfftHandle gchForward;

// calibration mask
static int gnCalibrationStart;
static int gnCalibrationStop;
static int gnCalibrationRound;
static float* gpfCalibrationMask;

// reverse fft
static hipfftComplex* gpcProcessSpectrum;
static size_t gnProcessSpectrumPitch;
static hipfftHandle gchReverse;

// phase
static float* gpfProcessPhase;
static size_t gnProcessPhasePitch;

// unwrap
static float gfPiEps = (float)(acos(-1.0) - 1.0e-30);
static float gf2Pi = (float)(2.0 * acos(-1.0));

// linear fit and interpolation
static float* gpfLeftPhase;
static float* gpfRightPhase;
static float* gpfKLineCoefficients;
static float* gpfProcessK;
static size_t gnKPitch;
static int* gpnProcessIndex;
static size_t gnIndexPitch;
static int* gpnProcessAssigned;
static size_t gnAssignedPitch;
static int gnKMode;
static float* gpfProcessSpectrumK;
static size_t gnSpectrumKPitch;

static float* gpfProcessOCT;
static size_t gnProcessOCTPitch;
static hipfftComplex* gpcProcessedOCT;

// dispersion mask
static int gnDispersionStart;
static int gnDispersionStop;
static int gnDispersionRound;
static float* gpfDispersionMask;

// dispersion correction
static hipfftComplex* gpcDispersionCorrection;
static hipfftHandle gchForwardComplex;
static hipfftComplex* gpcProcessKCorrected;
static size_t gnKCorrectedPitch;


int getDeviceCount(int* nNumberDevices) {
	
    // check for GPU
	int nDevices = -1; 
	int nRet = hipGetDeviceCount(&nDevices); 
	if (nRet == hipSuccess)
		*(nNumberDevices) = nDevices; 
	return nRet; 

}

int getDeviceName(int nDeviceNumber, char* strDeviceName) {
    
    // check for GPU
    hipDeviceProp_t currentDevice;
    int nRet = hipGetDeviceProperties(&currentDevice, nDeviceNumber);
    if (nRet == hipSuccess) {
        sprintf(strDeviceName, "%s (%d SMs, %d b/s, %d t/b, %d t/s, %d shared kB, %d GB)",
            currentDevice.name,
            currentDevice.multiProcessorCount,
            currentDevice.maxBlocksPerMultiProcessor,
            currentDevice.maxThreadsPerBlock,
            currentDevice.maxThreadsPerMultiProcessor,
            currentDevice.sharedMemPerBlock / 1024,
            currentDevice.totalGlobalMem / 1024 / 1024 / 1024);

    }	// if (nRet
    return nRet; 

}

int initialize(int nMode, int nRawLineLength, int nRawNumberLines, int nProcessNumberLines, int nProcessedNumberLines) {
    
    cleanup();

    // copy parameters to global parameters
    gnMode = nMode;
    gnRawLineLength = nRawLineLength;
    gnRawNumberLines = nRawNumberLines;
    gnProcessNumberLines = nProcessNumberLines;
    gnProcessedNumberLines = nProcessedNumberLines;

    // allocate memory
    switch (nMode) {
    case 0: // SD-OCT
        gnPerpendicular = 0;
        gnCalibrationNumberLines = 1;
        break;
    case 1: // PS SD-OCT
        gnPerpendicular = 1;
        gnCalibrationNumberLines = gnRawNumberLines;
        break;
    case 2: // line field
        gnPerpendicular = 0;
        gnCalibrationNumberLines = 1;
        break;
    case 3: // OFDI
        gnPerpendicular = 0;
        gnCalibrationNumberLines = gnRawNumberLines;
        break;
    case 4: // PS OFDI
        gnPerpendicular = 1;
        gnCalibrationNumberLines = gnRawNumberLines;
        break;
    } // switch (nMode)

    gpuErrchk(hipHostMalloc((void**)&gpfRawCalibration, (gnRawLineLength * gnCalibrationNumberLines) * sizeof(float)));
    gpuErrchk(hipMallocPitch((void**)&gpfProcessCalibration, &gnProcessCalibrationPitch, gnRawLineLength * sizeof(float), gnProcessNumberLines >> 1));

    gpuErrchk(hipMalloc((void**)&gpfReferenceEven, gnRawLineLength * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&gpfReferenceOdd, gnRawLineLength * sizeof(float)));

    gnMidLength = (int)(gnRawLineLength / 2 + 1);
    gpuErrchk(hipMallocPitch((void**)&gpcProcessDepthProfile, &gnProcessDepthProfilePitch, gnRawLineLength * sizeof(hipfftComplex), gnProcessNumberLines >> 1));
    int nRank = 1;
    int pn[] = { gnRawLineLength };
    int nIStride = 1, nOStride = 1;
    int nIDist = gnProcessCalibrationPitch / sizeof(float);
    int nODist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    int pnINEmbed[] = { 0 };
    int pnONEmbed[] = { 0 };
    int nBatch = gnProcessNumberLines >> 1;
    hipfftPlanMany(&gchForward, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_R2C, nBatch);

    gpuErrchk(hipMalloc((void**)&gpfCalibrationMask, gnRawLineLength * sizeof(float)));

    gpuErrchk(hipMallocPitch((void**)&gpcProcessSpectrum, &gnProcessSpectrumPitch, gnRawLineLength * sizeof(hipfftComplex), gnProcessNumberLines >> 1));
    nIDist = gnProcessDepthProfilePitch / sizeof(hipfftComplex);
    nODist = gnProcessSpectrumPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchReverse, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gpuErrchk(hipMallocPitch((void**)&gpfProcessPhase, &gnProcessPhasePitch, gnRawLineLength * sizeof(float), gnProcessNumberLines >> 1));

    hipMalloc((void**)&gpfLeftPhase, sizeof(float));
    hipMalloc((void**)&gpfRightPhase, sizeof(float));
    hipMalloc((void**)&gpfKLineCoefficients, 2 * sizeof(float));
    gpuErrchk(hipMallocPitch((void**)&gpfProcessK, &gnKPitch, gnRawLineLength * sizeof(float), gnProcessNumberLines >> 1));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessIndex, &gnIndexPitch, gnRawLineLength * sizeof(int), gnProcessNumberLines >> 1));
    gpuErrchk(hipMallocPitch((void**)&gpnProcessAssigned, &gnAssignedPitch, gnRawLineLength * sizeof(int), gnProcessNumberLines >> 1));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessSpectrumK, &gnSpectrumKPitch, gnRawLineLength * sizeof(float), gnProcessNumberLines >> 1));

    gpuErrchk(hipMallocPitch((void**)&gpfProcessOCT, &gnProcessOCTPitch, gnRawLineLength * sizeof(float), gnProcessNumberLines >> 1));
    gpuErrchk(hipHostMalloc((void**)&gpcProcessedOCT, (gnMidLength * gnProcessedNumberLines) * sizeof(hipfftComplex)));

    gpuErrchk(hipMalloc((void**)&gpfDispersionMask, gnRawLineLength * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&gpcDispersionCorrection, gnRawLineLength * sizeof(hipfftComplex)));
    gpuErrchk(hipMallocPitch((void**)&gpcProcessKCorrected, &gnKCorrectedPitch, gnRawLineLength * sizeof(hipfftComplex), gnProcessNumberLines >> 1));

    nIDist = gnKCorrectedPitch / sizeof(hipfftComplex);
    hipfftPlanMany(&gchForwardComplex, nRank, pn, pnINEmbed, nIStride, nIDist, pnONEmbed, nOStride, nODist, HIPFFT_C2C, nBatch);

    gnAllocationStatus = 1;

    return -1;

} // int initialize

int cleanup() {

    // free memory allocations
    if (gnAllocationStatus == 1) {
        gpuErrchk(hipHostFree(gpfRawCalibration));
        gpuErrchk(hipFree(gpfProcessCalibration));
        gpuErrchk(hipFree(gpfReferenceEven));
        gpuErrchk(hipFree(gpfReferenceOdd));
        gpuErrchk(hipFree(gpcProcessDepthProfile));
        hipfftDestroy(gchForward);
        gpuErrchk(hipFree(gpfCalibrationMask));
        gpuErrchk(hipFree(gpcProcessSpectrum));
        hipfftDestroy(gchReverse);
        gpuErrchk(hipFree(gpfProcessPhase));
        hipFree(gpfLeftPhase);
        hipFree(gpfRightPhase);
        hipFree(gpfKLineCoefficients);
        hipFree(gpfProcessK);
        hipFree(gpnProcessIndex);
        hipFree(gpnProcessAssigned);
        hipFree(gpfProcessSpectrumK);
        hipFree(gpfProcessOCT);
        hipHostFree(gpcProcessedOCT);
        gpuErrchk(hipFree(gpfDispersionMask));
        gpuErrchk(hipFree(gpcDispersionCorrection));
        hipfftDestroy(gchForwardComplex);
        hipFree(gpcProcessKCorrected);

        gnAllocationStatus = 0;
    }   // if (gnAllocationStatus
    return -1;

}

int getDataSDOCT(void* pnIMAQ) {
    return -1; }

int getDataPSSDOCT(void* pnIMAQParallel, void* pnIMAQPerpendicular) {
    return -1; 
}



int processPSSDOCT() {

}