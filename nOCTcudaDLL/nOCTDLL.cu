
#include "nOCTcudaDLLHeader.cuh"

int getDeviceCount(int* nNumberDevices) {
	// check for GPU
	int nDevices = -1; 
	int nRet = hipGetDeviceCount(&nDevices); 
	if (nRet == hipSuccess)
		*(nNumberDevices) = nDevices; 
	return nRet; 
}